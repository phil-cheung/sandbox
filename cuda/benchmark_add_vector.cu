#include "hip/hip_runtime.h"
/* This code benchmarks vector addition
#include <stdio.h>
#include <unistd.h>
#include <time.h>

const int BLOCKSIZE = 128; 

__global__
void vecAddKernel(float* A, float* B, float* C, int n) {
    int i = threadIdx.x + blockDim.x * blockIdx.x;
    if (i < n) C[i] = A[i] + B[i];
}


void vecAdd(float* A, float* B, float* C, int n) {
    int size = n * sizeof(float);
    int i;
    float *d_A, *d_B, *d_C;

    hipMalloc((void **) &d_A, size);
    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_B, size);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);
    hipMalloc((void **) &d_C, size);

    //Call Kernel function to add vector
    time_t a = time(NULL);
    for (i=0; i<100000000; i++)
        vecAddKernel<<<ceil(1.0*n/BLOCKSIZE), BLOCKSIZE>>>(d_A, d_B, d_C, n);
    time_t b = time(NULL);
    printf("Total time for all vector add iterations: %ld\n", b - a);


    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A); hipFree(d_B); hipFree(d_C);
}


int main () {
    float A[5] = {1, 2, 3, 4, 5};
    float B[5] = {5, 4, 3, 2, 1};
    float C[5];

    vecAdd(A, B, C, 5);

}
